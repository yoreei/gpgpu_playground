﻿

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>


// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
#define BLOCK_SIZE 2

typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;
// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}
// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}
// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    int offset = A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col;
    Asub.elements = &A.elements[offset];
    //printf("GetSubMatrix row: %d, col: %d, offset: %d\n", row, col, offset);
    return Asub;
}
// Thread block size

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);
    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);
    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}
// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;
    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;
    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        
        printf(
            "Thread [%d, %d] in Block [%d, %d]:\n"
            "  - Asub [blockRow: %d, m: %d]\n"
            "  - Bsub [m: %d, blockCol: %d]\n",
            row, col,            // Thread indices within the block
            blockRow, blockCol,  // Block indices within the grid
            blockRow, m,         // Location of Asub
            m, blockCol          // Location of Bsub
        );

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();

        if(row == 0 && col == 0){
            printf("__syncthreads\n");
        }
    }
    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

// Assuming Matrix struct and MatMul function are defined as in previous examples

// Function to initialize matrix with random data
void InitializeMatrix(Matrix &mat, int width, int height) {
    mat.width = width;
    mat.height = height;
    mat.stride = width; // Assuming stride equals width for simplicity
    size_t size = width * height * sizeof(float);
    mat.elements = (float*)malloc(size);

    for (int i = 0; i < width * height; i++) {
        mat.elements[i] = rand() % 100; // Random values between 0 and 99
    }
}

// Function to free matrix data
void FreeMatrix(Matrix &mat) {
    free(mat.elements);
}

// Main function
int main() {
    const int width = 4; // Assuming square matrices for simplicity
    const int height = 4;

    Matrix A, B, C;

    // Initialize matrices A and B with random data
    InitializeMatrix(A, width, height);
    InitializeMatrix(B, width, height);

    // Initialize matrix C (the result matrix) with zeros
    InitializeMatrix(C, width, height);
    for (int i = 0; i < width * height; i++) {
        C.elements[i] = 0;
    }

    // Perform matrix multiplication
    MatMul(A, B, C);

    // Assuming you have a function to print matrix for verification
    // PrintMatrix(C);

    // Free matrices
    FreeMatrix(A);
    FreeMatrix(B);
    FreeMatrix(C);

    std::cout << "Matrix multiplication completed successfully.\n";

    return 0;
}