﻿#include "matmul.cuh"
#include "histogram.cuh"
#include <iostream>
#include <hip/hip_runtime.h>

void printDeviceInfo()
{
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    }
}

int main() {
    printDeviceInfo();
    matmul::main();
}